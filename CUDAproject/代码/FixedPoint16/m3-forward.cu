#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <hip/hip_fp16.h>


#define TILE_WIDTH 16
#define BLOCK_SIZE 256

__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // const int unrolled_height = Channel * K * K;
    const size_t unrolled_width = Batch * Height_out * Width_out;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // __shared__ float shared_in[TILE_WIDTH + K - 1][TILE_WIDTH + K - 1];

    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define out_2d(i1, i2) output[(i1) * unrolled_width + (i2)]
    // TODO: Insert your input matrix unrolling kernel code here

    int batch_idx = blockIdx.z;

    // Calculate the output position
    int row_o = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col_o = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // Check if within output bounds
    if (row_o < Height_out && col_o < Width_out) {
        // Calculate the flat index for spatial position in output's width
        size_t w_o = row_o * Width_out + col_o + batch_idx * Height_out * Width_out;
        size_t h_o;
        // Unroll the KxK region and concatenate channels along rows
        for (int c = 0; c < Channel; ++c) {
            int idx = 0;
            for (int i = 0; i < K; ++i) {
                for (int j = 0; j < K; ++j) {
                    int row_in = row_o + i;
                    int col_in = col_o + j;
                    h_o = c * K * K + idx;
                    out_2d(h_o, w_o) = in_4d(batch_idx, c, row_in, col_in);
                    idx++;
                }
            }
        }
    }

    #undef in_4d
    #undef out_2d
}

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.

__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ __half tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ __half tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;
    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;

    // Initialize result as half-precision
    __half val = __float2half(0.0f);

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        // Load tiles as half-precision
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = __float2half(A[(size_t)row * numAColumns + tileId * TILE_WIDTH + tx]);
        } else {
            tileA[ty][tx] = __float2half(0.0f);
        }

        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = __float2half(B[(size_t)(tileId * TILE_WIDTH + ty) * numBColumns + col]);
        } else {
            tileB[ty][tx] = __float2half(0.0f);
        }

        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val = __hfma(tileA[ty][i], tileB[i][tx], val);
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = __half2float(val);  // Store result as float
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    // Allocate memory
    hipMalloc((void**)device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void**)device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)device_mask_ptr, Map_out * Channel * K * K * sizeof(float));

    // Copy data from host to device
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);
    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Height_unrolled = Channel * K * K;
    const int Width_unrolled = Batch * Height_out * Width_out;

    float *unrolled_matrix;  // Pointer to device memory for storing the unrolled matrix
    float *matmul_output;    // Pointer to device memory for storing the result of matrix multiplication
    hipMalloc((void**)&unrolled_matrix, (size_t) Batch * Channel * K * K * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)&matmul_output, (Batch * Map_out * Height_out * Width_out) * sizeof(float));

    // TODO: Set the kernel dimensions and call the matrix unrolling kernel.
    dim3 GridDim1((Width_out + TILE_WIDTH - 1) / TILE_WIDTH, (Height_out + TILE_WIDTH - 1) / TILE_WIDTH, Batch);
    dim3 BlockDim1(TILE_WIDTH, TILE_WIDTH);
    matrix_unrolling_kernel<<<GridDim1, BlockDim1>>>(device_input, unrolled_matrix, Batch, Channel, Height, Width, K);

    // TODO: Set the kernel dimensions and call the matmul kernel
    dim3 GridDim2((Width_unrolled + TILE_WIDTH - 1) / TILE_WIDTH, (Map_out + TILE_WIDTH - 1) / TILE_WIDTH);
    dim3 BlockDim2(TILE_WIDTH, TILE_WIDTH);

    // Call the matrix multiplication kernel
    matrixMultiplyShared<<<GridDim2, BlockDim2>>>(
        device_mask, unrolled_matrix, matmul_output, Map_out, Height_unrolled, Height_unrolled, Width_unrolled, Map_out, Width_unrolled);   
    // Permute the result of matrix multiplication
    const int out_image_size = Height_out * Width_out;
    dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, Batch, 1);
    matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE>>>(
        matmul_output, device_output, Map_out, Batch, out_image_size
    );

    hipFree(matmul_output);
    hipFree(unrolled_matrix);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;    
    hipMemcpy(host_output, device_output, Batch * Map_out * Height_out * Width_out * sizeof(float), hipMemcpyDeviceToHost);
    // TODO: Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}