#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256

__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // const int unrolled_height = Channel * K * K;
    const size_t unrolled_width = Batch * Height_out * Width_out;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // __shared__ float shared_in[TILE_WIDTH + K - 1][TILE_WIDTH + K - 1];

    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define out_2d(i1, i2) output[(i1) * unrolled_width + (i2)]
    // TODO: Insert your input matrix unrolling kernel code here

    int batch_idx = blockIdx.z;

    // Calculate the output position
    int row_o = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col_o = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // Check if within output bounds
    if (row_o < Height_out && col_o < Width_out) {
        // Calculate the flat index for spatial position in output's width
        size_t w_o = row_o * Width_out + col_o + batch_idx * Height_out * Width_out;
        size_t h_o;
        // Unroll the KxK region and concatenate channels along rows
        for (int c = 0; c < Channel; ++c) {
            int idx = 0;
            for (int i = 0; i < K; ++i) {
                for (int j = 0; j < K; ++j) {
                    int row_in = row_o + i;
                    int col_in = col_o + j;
                    h_o = c * K * K + idx;
                    out_2d(h_o, w_o) = in_4d(batch_idx, c, row_in, col_in);
                    idx++;
                }
            }
        }
    }

    #undef in_4d
    #undef out_2d
}


// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

// __host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
// {
//     // TODO: Allocate memory and copy over the relevant data structures to the GPU

//     // We pass double pointers for you to initialize the relevant device pointers,
//     //  which are passed to the other two functions.
//     const int Height_out = Height - K + 1;
//     const int Width_out = Width - K + 1;
//     //const to new pointers
//     // float* ptr = (float*) host_output;
//     // Allocate memory
//     hipMalloc((void**)device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
//     hipMalloc((void**)device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
//     hipMalloc((void**)device_mask_ptr, Map_out * Channel * K * K * sizeof(float));

//     // Copy data from host to device
//     hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
//     hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);
//     // Useful snippet for error checking
//     // hipError_t error = hipGetLastError();
//     // if(error != hipSuccess)
//     // {
//     //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
//     //     exit(-1);
//     // }

// }

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Height_unrolled = Channel * K * K;
    const int Width_unrolled = Batch * Height_out * Width_out;

    // Allocate pinned memory for host data
    float *host_input_pinned, *host_output_pinned;
    hipHostMalloc((void**)&host_input_pinned, Batch * Channel * Height * Width * sizeof(float));
    hipHostMalloc((void**)&host_output_pinned, Batch * Map_out * Height_out * Width_out * sizeof(float));

    // Allocate device memory
    hipMalloc((void**)device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void**)device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)device_mask_ptr, Map_out * Channel * K * K * sizeof(float));
    float  *unrolled_matrix, *matmul_output;
    hipMalloc((void**)&unrolled_matrix, (size_t)Batch * Channel * K * K * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)&matmul_output, Batch * Map_out * Height_out * Width_out * sizeof(float));

    // Copy data from existing host pointers into pinned memory
    memcpy(host_input_pinned, host_input, Batch * Channel * Height * Width * sizeof(float));
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);

    // Number of streams
    const int nStreams = 5; 
    hipStream_t streams[nStreams];
    for (int i = 0; i < nStreams; ++i) {
        hipStreamCreate(&streams[i]);
    }
    int streamSize_thread =  Batch * Channel * Height_out * Width_out / nStreams;
    int streamSize_input = Batch * Channel * Height * Width / nStreams;
    size_t streamSize_unrolled = (size_t)Batch * Channel * K * K * Height_out * Width_out / nStreams;
    int streamSize_output = Batch * Map_out * Height_out * Width_out / nStreams;
    int streamByteCpy = Batch * Channel * Height * Width * sizeof(float) / nStreams;
    int streamByteCpy2 = Batch * Map_out * Height_out * Width_out * sizeof(float) / nStreams;
    for (int i = 0; i < nStreams; ++i){
        int offset_input = i * streamSize_input;
        size_t offset_unrolled = i * streamSize_unrolled;
        int offset_output = i * streamSize_output;
    // Copy data from host to device
    hipMemcpyAsync(*device_input_ptr + offset_input, host_input_pinned + offset_input, streamByteCpy , hipMemcpyHostToDevice, streams[i]);
    }
    for (int i = 0; i < nStreams; ++i){
    // Launch the matrix unrolling kernel
    int offset_input = i * streamSize_input;
    size_t offset_unrolled = i * streamSize_unrolled;
    int offset_output = i * streamSize_output;
    int num_threads = streamSize_thread;
    int num_blocks = (num_threads + BLOCK_SIZE - 1) / BLOCK_SIZE;
    // dim3 GridDim1(num_blocks, 1, 1);
    // dim3 BlockDim1(BLOCK_SIZE, 1, 1);
    dim3 GridDim1((Width_out + TILE_WIDTH - 1) / TILE_WIDTH, (Height_out + TILE_WIDTH - 1) / TILE_WIDTH, Batch / nStreams);
    dim3 BlockDim1(TILE_WIDTH, TILE_WIDTH);
    matrix_unrolling_kernel<<<GridDim1, BlockDim1, 0, streams[i]>>>((*device_input_ptr + offset_input), unrolled_matrix + offset_unrolled, Batch / nStreams, Channel, Height, Width, K);

    // Launch the matrix multiplication kernel
    dim3 GridDim2((Width_unrolled / nStreams + TILE_WIDTH - 1) / TILE_WIDTH, (Map_out + TILE_WIDTH - 1) / TILE_WIDTH);
    dim3 BlockDim2(TILE_WIDTH, TILE_WIDTH);
    matrixMultiplyShared<<<GridDim2, BlockDim2, 0, streams[i]>>>(
        (*device_mask_ptr), unrolled_matrix + offset_unrolled, matmul_output + offset_output, Map_out, Height_unrolled, Height_unrolled, Width_unrolled / nStreams, Map_out, Width_unrolled / nStreams);

    // Permute the result of matrix multiplication
    const int out_image_size = Height_out * Width_out;
    dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, Batch/nStreams, 1);
    matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE,0,streams[i]>>>(
        matmul_output + offset_output, (*device_output_ptr) + offset_output, Map_out, Batch/nStreams, out_image_size
    );
    }
    for (int i = 0; i < nStreams; ++i){
    // Copy the output back to host
    int offset_input = i * streamSize_input;
    size_t offset_unrolled = i * streamSize_unrolled;
    int offset_output = i * streamSize_output;
    hipMemcpyAsync(((float*)host_output_pinned + offset_output), (*device_output_ptr) + offset_output, streamByteCpy2, hipMemcpyDeviceToHost, streams[i]);
    }
    //Synchronize streams and destory
    for (int i = 0; i < nStreams; ++i) {
        hipStreamSynchronize(streams[i]);
        hipStreamDestroy(streams[i]);
    }
    hipMemcpy((float*)host_output,host_output_pinned, Batch * Map_out * Height_out * Width_out * sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(*device_input_ptr);
    hipFree(*device_mask_ptr);
    hipFree(*device_output_ptr);
    hipFree(unrolled_matrix);
    hipFree(matmul_output);
    hipFree(host_input_pinned);
    hipFree(host_output_pinned);
    // hipFree(host_mask_pinned);

}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // const int Height_out = Height - K + 1;
    // const int Width_out = Width - K + 1;
    // const int Height_unrolled = Channel * K * K;
    // const int Width_unrolled = Batch * Height_out * Width_out;

    // float *unrolled_matrix;  // Pointer to device memory for storing the unrolled matrix
    // float *matmul_output;    // niPointer to device memory for storing the result of matrix multiplication
    // hipMalloc((void**)&unrolled_matrix, (size_t) Batch * Channel * K * K * Height_out * Width_out * sizeof(float));
    // hipMalloc((void**)&matmul_output, (Batch * Map_out * Height_out * Width_out) * sizeof(float));

    // // TODO: Set the kernel dimensions and call the matrix unrolling kernel.
    // int num_threads = Batch * Channel * Height_out * Width_out;
    // int num_blocks = ceil((num_threads) / BLOCK_SIZE);
    // // dim3 GridDim1((Width_out + TILE_WIDTH - 1) / TILE_WIDTH, (Height_out + TILE_WIDTH - 1) / TILE_WIDTH, Batch);
    // dim3 GridDim1(num_blocks,1,1);
    // dim3 BlockDim1(BLOCK_SIZE,1,1);
    // matrix_unrolling_kernel<<<GridDim1, BlockDim1>>>(device_input, unrolled_matrix, Batch, Channel, Height, Width, K);

    // // TODO: Set the kernel dimensions and call the matmul kernel
    // dim3 GridDim2((Width_unrolled + TILE_WIDTH - 1) / TILE_WIDTH, (Map_out + TILE_WIDTH - 1) / TILE_WIDTH);
    // dim3 BlockDim2(TILE_WIDTH, TILE_WIDTH);

    // // Call the matrix multiplication kernel
    // matrixMultiplyShared<<<GridDim2, BlockDim2>>>(
    //     device_mask, unrolled_matrix, matmul_output, Map_out, Height_unrolled, Height_unrolled, Width_unrolled, Map_out, Width_unrolled);   
    // // Permute the result of matrix multiplication
    // const int out_image_size = Height_out * Width_out;
    // dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, Batch, 1);
    // matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE>>>(
    //     matmul_output, device_output, Map_out, Batch, out_image_size
    // );

    // hipFree(matmul_output);
    // hipFree(unrolled_matrix);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // // TODO: Copy the output back to host
    // const int Height_out = Height - K + 1;
    // const int Width_out = Width - K + 1;    
    // hipMemcpy(host_output, device_output, Batch * Map_out * Height_out * Width_out * sizeof(float), hipMemcpyDeviceToHost);
    // // TODO: Free device memory
    // hipFree(device_output);
    // hipFree(device_input);
    // hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}