#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include <mma.h> 
#define __CUDA_ARCH__ 1180

#define BLOCK_TENSOR_X 32
#define BLOCK_TENSOR_Y 32
#define TILE_WIDTH 16
#define BLOCK_SIZE 256
#define WMMA_M 16
#define WMMA_N 16
#define WMMA_K 16
#define WARP_SIZE 32

const int warpNum = BLOCK_TENSOR_X * BLOCK_TENSOR_Y / WARP_SIZE;
const int warpY = 8;
const int warpX = warpNum / warpY;

using namespace nvcuda;

__global__ void matrix_unrolling_kernel(const float *input, float *output,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // const int unrolled_height = Channel * K * K;
    const size_t unrolled_width = Batch * Height_out * Width_out;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // __shared__ float shared_in[TILE_WIDTH + K - 1][TILE_WIDTH + K - 1];

    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define out_2d(i1, i2) output[(i1) * unrolled_width + (i2)]
    // TODO: Insert your input matrix unrolling kernel code here

    int batch_idx = blockIdx.z;

    // Calculate the output position
    int row_o = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col_o = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // Check if within output bounds
    if (row_o < Height_out && col_o < Width_out) {
        // Calculate the flat index for spatial position in output's width
        size_t w_o = row_o * Width_out + col_o + batch_idx * Height_out * Width_out;
        size_t h_o;
        // Unroll the KxK region and concatenate channels along rows
        for (int c = 0; c < Channel; ++c) {
            int idx = 0;
            for (int i = 0; i < K; ++i) {
                for (int j = 0; j < K; ++j) {
                    int row_in = row_o + i;
                    int col_in = col_o + j;
                    h_o = c * K * K + idx;
                    out_2d(h_o, w_o) = in_4d(batch_idx, c, row_in, col_in);
                    idx++;
                }
            }
        }
    }

    #undef in_4d
    #undef out_2d
}


// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const half *A, const half *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float *input, float *output, int Map_out,
                                      int Batch, int image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}



__global__ void matrixMultiplyTensorCore(half *dA, half *dB, float *dC, size_t M, size_t K, size_t N)
{
    int lda = K;
    int ldb = N;
    int ldc = N;

    // size_t indA = blockIdx.x * warpX * WMMA_M;
    // size_t indB = blockIdx.y * warpY * WMMA_N;

    int warpId = (threadIdx.x + threadIdx.y * blockDim.x) / WARP_SIZE;
    int warpIdx = warpId % warpX;
    int warpIdy = warpId / warpX;
    // size_t warp_M = (blockIdx.x * blockDim.x + threadIdx.x)/32;
    // size_t warp_N = (blockIdx.y * blockIdx.y) + threadIdx.y;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K,half, wmma::row_major> frag_A;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> frag_B;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> frag_C;

    wmma::fill_fragment(frag_C, 0.0f);
    size_t aRow = blockIdx.y * warpY * WMMA_M + warpIdy * WMMA_M;
    size_t bCol = blockIdx.x * warpX * WMMA_N + warpIdx * WMMA_N;
    // size_t aRow = warp_M * WMMA_M;
    // size_t bCol = warp_N * WMMA_N;
    int width = (K + WMMA_K - 1) / WMMA_K;
    for (int i = 0; i < width; i++)
    {
        int aCol = i * WMMA_K;
        size_t bRow = i * WMMA_K;
        if (aRow < M && aCol < K && bRow < K && bCol < N)
        {
            wmma::load_matrix_sync(frag_A, dA + (size_t)(aRow * lda) + (size_t)aCol, lda);
            wmma::load_matrix_sync(frag_B, dB + (size_t)(bRow * ldb) + (size_t)bCol, ldb);
            wmma::mma_sync(frag_C, frag_A, frag_B, frag_C);
        }
    }
    size_t cRow = aRow;
    size_t cCol = bCol;
    if (cRow < M && cCol < N)
    {
        wmma::store_matrix_sync(dC + (size_t)(cRow * ldc) + (size_t)cCol, frag_C, ldc, wmma::mem_row_major);
    }
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    //const to new pointers
    // float* ptr = (float*) host_output;
    // Allocate memory
    hipMalloc((void**)device_input_ptr, Batch * Channel * Height * Width * sizeof(float));
    hipMalloc((void**)device_output_ptr, Batch * Map_out * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)device_mask_ptr, Map_out * Channel * K * K * sizeof(float));

    // Copy data from host to device
    hipMemcpy(*device_input_ptr, host_input, Batch * Channel * Height * Width * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, Map_out * Channel * K * K * sizeof(float), hipMemcpyHostToDevice);

}

//helper function to pad matrices and convert the datatype to half
__global__ void padMatrixToHalfKernel(const float* input, __half* output, size_t rows, size_t cols, size_t padded_rows, size_t padded_cols) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    size_t y = blockIdx.y * blockDim.y + threadIdx.y; // Row index

    if (x < padded_cols && y < padded_rows) {
        if (x < cols && y < rows) {
            // Copy and convert float to half
            output[y * padded_cols + x] = __float2half(input[y * cols + x]);
        } else {
            // Fill padding with 0 in half precision
            output[y * padded_cols + x] = __float2half(0.0f);
        }
    }
}

__global__ void removePadding(float *paddedMatrix, float *originalMatrix, 
                              size_t paddedRows, size_t paddedCols, 
                              size_t originalRows, size_t originalCols) {
    // Calculate the row and column index in the original matrix
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the current thread corresponds to a valid position in the original matrix
    if (row < originalRows && col < originalCols) {
        // Map the element from the padded matrix to the original matrix
        originalMatrix[row * originalCols + col] = paddedMatrix[row * paddedCols + col];
    }
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, 
                                             const int Batch, const int Map_out, const int Channel, 
                                             const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const size_t Height_unrolled = Channel * K * K;
    const size_t Width_unrolled = Batch * Height_out * Width_out;

    float *unrolled_matrix;  // Pointer to device memory for unrolled matrix
    float *matmul_output;    // Pointer to device memory for result
    hipMalloc(&unrolled_matrix, (size_t)Batch * Channel * K * K * Height_out * Width_out * sizeof(float));
    hipMalloc(&matmul_output, Batch * Map_out * Height_out * Width_out * sizeof(float));

    // Unroll input
    dim3 GridDim1((Width_out + TILE_WIDTH - 1) / TILE_WIDTH, (Height_out + TILE_WIDTH - 1) / TILE_WIDTH, Batch);
    dim3 BlockDim1(TILE_WIDTH, TILE_WIDTH);
    matrix_unrolling_kernel<<<GridDim1, BlockDim1>>>(device_input, unrolled_matrix, Batch, Channel, Height, Width, K);

    // Allocate padded matrices
    size_t paddedARows = (Map_out + TILE_WIDTH - 1) / TILE_WIDTH * TILE_WIDTH;
    size_t paddedACols = (Height_unrolled + TILE_WIDTH - 1) / TILE_WIDTH * TILE_WIDTH;
    size_t paddedBCols = (Width_unrolled + TILE_WIDTH - 1) / TILE_WIDTH * TILE_WIDTH;

    half *device_mask_padded, *unrolled_matrix_padded;
    hipMalloc(&unrolled_matrix_padded, (size_t)paddedACols * paddedBCols * sizeof(half));
    hipMalloc(&device_mask_padded, (size_t)paddedARows * paddedACols * sizeof(half));

    // Pad device mask
    dim3 blockDim2(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim2((paddedACols + TILE_WIDTH - 1) / TILE_WIDTH, (paddedARows + TILE_WIDTH - 1) / TILE_WIDTH);
    padMatrixToHalfKernel<<<gridDim2, blockDim2>>>(device_mask, device_mask_padded, Map_out, Height_unrolled, paddedARows, paddedACols);
    hipDeviceSynchronize();

    // Pad unrolled matrix
    dim3 blockDim3(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim3((paddedBCols + TILE_WIDTH - 1) / TILE_WIDTH, (paddedACols + TILE_WIDTH - 1) / TILE_WIDTH);
    padMatrixToHalfKernel<<<gridDim3, blockDim3>>>(unrolled_matrix, unrolled_matrix_padded, Height_unrolled, Width_unrolled, paddedACols, paddedBCols);
    hipDeviceSynchronize();

    float* matmul_output_paddded;
    hipMalloc(&matmul_output_paddded, (size_t)paddedARows * paddedBCols * sizeof(float));
    // Tensor Core Matrix Multiplication
    dim3 blockDim4(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim4((paddedBCols + TILE_WIDTH - 1) / TILE_WIDTH, (paddedARows + TILE_WIDTH - 1) / TILE_WIDTH);   
    // matrixMultiplyTensorCore<<<gridDim4, blockDim4>>>(device_mask_padded, unrolled_matrix_padded, matmul_output_paddded,
    //                                                   paddedARows, paddedACols, paddedACols, paddedBCols, paddedARows, paddedBCols);
    // matrixMultiplyShared<<<gridDim4, blockDim4>>>(device_mask_padded, unrolled_matrix_padded, matmul_output_paddded,
    //                                             paddedARows, paddedACols, paddedACols, paddedBCols, paddedARows, paddedBCols);
// Configuration for WMMA
    // int num_block_x = (paddedARows + WMMA_M * warpX - 1) / (WMMA_M * warpX);
    // int num_block_y = (paddedBCols + WMMA_N * warpY - 1) / (WMMA_N * warpY);
    int num_block_x = (paddedBCols + WMMA_N * warpX - 1) / (WMMA_N * warpX);
    int num_block_y = (paddedARows + WMMA_M * warpY - 1) / (WMMA_M * warpY);

    dim3 block_dim(BLOCK_TENSOR_X, BLOCK_TENSOR_Y, 1);
    dim3 grid_dim(num_block_x, num_block_y, 1);
    // dim3 blockSize(32, 32, 1);
    // int block_num_x = (paddedARows / 16 + blockSize.x - 1) / blockSize.x;
    // int block_num_y = (paddedBCols / 16 + blockSize.y - 1) / blockSize.y;
    // dim3 gridSize(block_num_x, block_num_y, 1);
    matrixMultiplyTensorCore<<<grid_dim, block_dim>>>(device_mask_padded, unrolled_matrix_padded, matmul_output_paddded, paddedARows, paddedACols, paddedBCols);

    //remove padding drom padded matmult output
    dim3 blockDim(16, 16); 
    dim3 gridDim((Width_unrolled + blockDim.x - 1) / blockDim.x, 
                (Map_out + blockDim.y - 1) / blockDim.y);
    removePadding<<<gridDim, blockDim>>>(matmul_output_paddded, matmul_output, 
                                        paddedARows, paddedBCols, 
                                        Map_out, Width_unrolled);
    hipDeviceSynchronize();

    // Permute the result back
    const int out_image_size = Height_out * Width_out;
    dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, Batch, 1);
    matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE>>>(matmul_output, device_output, Map_out, Batch, out_image_size);
    hipDeviceSynchronize();

    // Free device memory
    hipFree(device_mask_padded);
    hipFree(unrolled_matrix_padded);
    hipFree(matmul_output);
    hipFree(unrolled_matrix);
    hipFree(matmul_output_paddded);
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;    
    hipMemcpy(host_output, device_output, Batch * Map_out * Height_out * Width_out * sizeof(float), hipMemcpyDeviceToHost);
    // TODO: Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}